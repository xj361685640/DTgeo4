
#include <hip/hip_runtime.h>
__global__ void kernel(float* a, const hipTextureObject_t* tex){
  a[0] = tex3D<float>(tex[blockIdx.x], 0.1, 0.2, 0.3);
}

