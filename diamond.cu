#include "hip/hip_runtime.h"
#include <stdio.h>
#include <omp.h>
#ifdef MPI_ON
#include <mpi.h>
#endif
#include "chooseV.h"
#include "signal.h"

int* mapNodeSize;
//=============================================
ftype* __restrict__ hostKpmlx1; ftype* __restrict__ hostKpmlx2;
ftype* __restrict__ hostKpmly1; ftype* __restrict__ hostKpmly2;
ftype* __restrict__ hostKpmlz1; ftype* __restrict__ hostKpmlz2;
GeoParamsHost parsHost;
__constant__ GeoParams pars;
__constant__ uint32_t drop_cells[Ns*NDT*Nwarps];
__constant__ int devNStripe[NDev] = STRIPES;
__constant__ ftype Kpmlx1[(KNpmlx==0)?1:KNpmlx];
__constant__ ftype Kpmlx2[(KNpmlx==0)?1:KNpmlx];
__constant__ ftype Kpmly1[(KNpmly==0)?1:KNpmly];
__constant__ ftype Kpmly2[(KNpmly==0)?1:KNpmly];
__constant__ ftype Kpmlz1[(KNpmlz==0)?1:KNpmlz];
__constant__ ftype Kpmlz2[(KNpmlz==0)?1:KNpmlz];
//__shared__ ftype2 shared_fld[2][7][Nz];
//__shared__ ftype2 shared_fld[(FTYPESIZE*Nv*28>0xc000)?7:14][Nv];
__shared__ ftype2 shared_fld[SHARED_SIZE][Nv];
texture<char, hipTextureType3D> index_tex;
hipArray* index_texArray=0;

#include "window.hpp"
#define IFPMLS(func,a,b,c,d,args) {/*printf(#func" idev=%d ix=%d iym=%d Nblocks=%d\n", idev,ix, iym, a);*/if(isPMLs) PMLS##func<<<a,b,c,d>>>args; else func<<<a,b,c,d>>>args; }
//#define IFPMLS(func,a,b,c,d,args) { if(!isPMLs) func<<<a,b,c,d>>>args; }
//#define IFPMLS(func,a,b,c,d,args) func<<<a,b,c,d>>>args;
template<int even> inline void Window::Dtorre(int ix, int Nt, int t0, double disbal[NDev], bool isPMLs, bool isTFSF) {
  if(Nt<=t0 || Nt<=0) return;
  DEBUG_PRINT(("Dtorre%d isPMLs=%d isTFSF=%d ix=%d, t0=%d Nt=%d wleft=%d\n", even, isPMLs, isTFSF, ix,t0,Nt, parsHost.wleft));
  const int Nth=Nv; 
  double tt1 = omp_get_wtime();
  CHECK_ERROR( hipSetDevice(0) );
  hipStream_t stPMLm; CHECK_ERROR( hipStreamCreate(&stPMLm) );
  hipStream_t stDm[NDev],stDo[NDev]; for(int i=0;i<NDev;i++) { if(i!=0) CHECK_ERROR( hipSetDevice(i) ); CHECK_ERROR( hipStreamCreate(&stDm[i]) ); CHECK_ERROR( hipStreamCreate(&stDo[i]) ); }
  hipStream_t stPMLp; CHECK_ERROR( hipStreamCreate(&stPMLp) );
  hipStream_t stX   ; CHECK_ERROR( hipStreamCreate(&stX   ) );
  hipStream_t stP   ; if(even==0) { hipSetDevice(NDev-1); CHECK_ERROR( hipStreamCreate(&stP   ) ); } else
                       if(even==1) { hipSetDevice(0     ); CHECK_ERROR( hipStreamCreate(&stP   ) ); }
  CHECK_ERROR( hipSetDevice(0) );

  int iym=0, iyp=0; 
  int Nblk=0;   iyp++;
  int Iy=iym, Xy, D1oy[NDev], D0oy[NDev], Dmy[NDev], DmBlk[NDev], Sy, SyBlk;
  int is_oneL[NDev], is_oneU[NDev], is_many[NDev], is_I[NDev], is_X[NDev], is_S[NDev], is_P[NDev];
  for(int i=0; i<NDev; i++) { is_oneL[i]=0; is_oneU[i]=0; is_many[i]=0; is_I[i]=0; is_X[i]=0; is_S[i]=0; is_P[i]=0; }
  is_I[0]=1;
  for(int idev=0,nextY=0; idev<NDev; idev++) {
    nextY+=NStripe[idev]; if(idev==NDev-1) nextY-=Npmly;
    if(idev!=0) {
    // Dtorre1 only
      if(iyp<nextY && even==1) is_oneL[idev]=1;
      D1oy[idev]=iyp; if(iyp<nextY) iyp++;
    }
    iym=iyp; Nblk=0;  while(iyp<nextY-(idev==NDev-1?0:1)) { iyp++; Nblk++; }
    // Main Region
    if(Nblk>0) is_many[idev]=1;
    Dmy[idev]=iym, DmBlk[idev]=Nblk;
    if(idev!=NDev-1) {
    // Dtorre0 only
      if(iyp<nextY && even==0) is_oneU[idev]=1;
      D0oy[idev]=iyp; if(iyp<nextY) iyp++;
    }
  }
  iym=iyp; Nblk=0;  while(iyp<Na-1) { iyp++; Nblk++; }
  if(Nblk>0) is_S[NDev-1]=1;
  is_X[NDev-1]=1;
  Sy=iym; SyBlk=Nblk; Xy=iyp;
  if(subnode!=0) {
    is_I[0]=0; is_P[0]=1;
  }
  if(subnode!=NasyncNodes-1) {
    is_X[NDev-1]=0; is_P[NDev-1]=1; 
    //TODO disable is_S for all except NDev-1///
  }

  for(int idev=0; idev<NDev; idev++) {
    if(idev!=0) CHECK_ERROR( hipSetDevice(idev) );
    if(is_oneL[idev] && even==1 &&  isTFSF) IFPMLS(torreTFSF1 ,1          ,Nth,0,stDo[idev],(ix,D1oy[idev],Nt,t0))
    if(is_oneL[idev] && even==1 && !isTFSF) IFPMLS(torreD1    ,1          ,Nth,0,stDo[idev],(ix,D1oy[idev],Nt,t0))
    if(is_oneU[idev] && even==0 &&  isTFSF) IFPMLS(torreTFSF0 ,1          ,Nth,0,stDo[idev],(ix,D0oy[idev],Nt,t0))
    if(is_oneU[idev] && even==0 && !isTFSF) IFPMLS(torreD0    ,1          ,Nth,0,stDo[idev],(ix,D0oy[idev],Nt,t0))
    if(is_I[idev]    && even==0 &&  isTFSF) IFPMLS(torreITFSF0,1          ,Nth,0,stPMLm    ,(ix,Iy        ,Nt,t0))
    if(is_I[idev]    && even==0 && !isTFSF) IFPMLS(torreI0    ,1          ,Nth,0,stPMLm    ,(ix,Iy        ,Nt,t0))
    if(is_I[idev]    && even==1 &&  isTFSF) IFPMLS(torreITFSF1,1          ,Nth,0,stPMLm    ,(ix,Iy        ,Nt,t0))
    if(is_I[idev]    && even==1 && !isTFSF) IFPMLS(torreI1    ,1          ,Nth,0,stPMLm    ,(ix,Iy        ,Nt,t0))
    if(is_X[idev]    && even==0           ) IFPMLS(torreX0    ,1          ,Nth,0,stX       ,(ix,Xy        ,Nt,t0))
    if(is_X[idev]    && even==1           ) IFPMLS(torreX1    ,1          ,Nth,0,stX       ,(ix,Xy        ,Nt,t0))
    if(is_P[idev]    && even==0           ) IFPMLS(torreD0    ,1          ,Nth,0,stP       ,(ix,Xy        ,Nt,t0))
    if(is_P[idev]    && even==1           ) IFPMLS(torreD1    ,1          ,Nth,0,stP       ,(ix,Iy        ,Nt,t0))
    if(is_S[idev]    && even==0           ) IFPMLS(torreS0    ,SyBlk      ,Nth,0,stPMLp    ,(ix,Sy        ,Nt,t0))
    if(is_S[idev]    && even==1           ) IFPMLS(torreS1    ,SyBlk      ,Nth,0,stPMLp    ,(ix,Sy        ,Nt,t0))
    if(is_many[idev] && even==0 && isTFSF ) IFPMLS(torreTFSF0 ,DmBlk[idev],Nth,0,stDm[idev],(ix,Dmy[idev] ,Nt,t0))
    if(is_many[idev] && even==1 && isTFSF ) IFPMLS(torreTFSF1 ,DmBlk[idev],Nth,0,stDm[idev],(ix,Dmy[idev] ,Nt,t0))
    if(is_many[idev] && even==0 && !isTFSF) IFPMLS(torreD0    ,DmBlk[idev],Nth,0,stDm[idev],(ix,Dmy[idev] ,Nt,t0))
    if(is_many[idev] && even==1 && !isTFSF) IFPMLS(torreD1    ,DmBlk[idev],Nth,0,stDm[idev],(ix,Dmy[idev] ,Nt,t0))
    if(is_oneL[idev] && even==1           ) for(int ixrag=ix; ixrag<ix+Nt-t0; ixrag++) DiamondRag::copyM(idev, ixrag, stDo[idev]);
    if(is_oneU[idev] && even==0           ) for(int ixrag=ix; ixrag<ix+Nt-t0; ixrag++) DiamondRag::copyP(idev, ixrag, stDo[idev]);
  }

  /*
  if(even==0                       ) IFPMLS(torreI0   ,1   ,Nth,0,stPMLm   ,(ix,iym,Nt,t0))
  if(even==1                       ) IFPMLS(torreI1   ,1   ,Nth,0,stPMLm   ,(ix,iym,Nt,t0))
  for(int idev=0,nextY=0; idev<NDev; idev++) {
    if(idev!=0) CHECK_ERROR( hipSetDevice(idev) );
    nextY+=NStripe[idev]; if(idev==NDev-1) nextY-=Npmly;
    if(idev!=0) { iym=iyp;
    if(iyp<nextY && even==1 && isTFSF ) IFPMLS(torreTFSF1,1,Nth,0,stDo[idev],(ix,iym,Nt,t0))
    if(iyp<nextY && even==1 && !isTFSF) IFPMLS(torreD1   ,1,Nth,0,stDo[idev],(ix,iym,Nt,t0))
    if(iyp<nextY && even==1 ) for(int ixrag=ix; ixrag<ix+Nt-t0; ixrag++) DiamondRag::copyM(idev, ixrag, stDo[idev]);
    if(iyp<nextY) iyp++;
    }
    iym=iyp; Nblk=0;  while(iyp<nextY-(idev==NDev-1?0:1)) { iyp++; Nblk++; }
    if(Nblk>0 && even==0 && isTFSF ) IFPMLS(torreTFSF0,Nblk,Nth,0,stDm[idev],(ix,iym,Nt,t0))
    if(Nblk>0 && even==1 && isTFSF ) IFPMLS(torreTFSF1,Nblk,Nth,0,stDm[idev],(ix,iym,Nt,t0))
    if(Nblk>0 && even==0 && !isTFSF) IFPMLS(torreD0   ,Nblk,Nth,0,stDm[idev],(ix,iym,Nt,t0))
    if(Nblk>0 && even==1 && !isTFSF) IFPMLS(torreD1   ,Nblk,Nth,0,stDm[idev],(ix,iym,Nt,t0))
    if(idev!=NDev-1) { iym=iyp;
    if(iyp<nextY && even==0 && isTFSF ) IFPMLS(torreTFSF0,1,Nth,0,stDo[idev],(ix,iym,Nt,t0))
    if(iyp<nextY && even==0 && !isTFSF) IFPMLS(torreD0   ,1,Nth,0,stDo[idev],(ix,iym,Nt,t0))
    if(iyp<nextY && even==0 ) for(int ixrag=ix; ixrag<ix+Nt-t0; ixrag++) DiamondRag::copyP(idev, ixrag, stDo[idev]);
    if(iyp<nextY) iyp++;
    }
  }
  iym=iyp; Nblk=0;  while(iyp<Na-1        ) { iyp++; Nblk++; } 
  if(Nblk>0 && even==0             ) IFPMLS(torreS0   ,Nblk,Nth,0,stPMLp   ,(ix,iym,Nt,t0))
  if(Nblk>0 && even==1             ) IFPMLS(torreS1   ,Nblk,Nth,0,stPMLp   ,(ix,iym,Nt,t0))
  if(even==0                       ) IFPMLS(torreX0   ,1   ,Nth,0,stX      ,(ix,iyp,Nt,t0))
  if(even==1                       ) IFPMLS(torreX1   ,1   ,Nth,0,stX      ,(ix,iyp,Nt,t0))*/
  
  CHECK_ERROR( hipSetDevice(0) );

  if(!doneMemcopy) {
    if(even==0) MemcopyDtH(ix4copy);
    if(even==1) MemcopyHtD(ix4copy);
    CHECK_ERROR( hipStreamSynchronize(streamCopy) ); if(even==1) doneMemcopy=true;
  }
  if(even==1) parsHost.drop.save(stPMLm);
  CHECK_ERROR( hipStreamSynchronize(stPMLm) ); 
  CHECK_ERROR( hipStreamSynchronize(stPMLp) );
  CHECK_ERROR( hipStreamSynchronize(stX   ) );
  CHECK_ERROR( hipStreamSynchronize(stP   ) );
  for(int i=0;i<NDev;i++) CHECK_ERROR( hipStreamSynchronize(stDo[i]) );
  for(int i=0;i<NDev;i++) { double tt=omp_get_wtime(); CHECK_ERROR( hipStreamSynchronize(stDm[i]) ); disbal[i]+=omp_get_wtime()-tt; }
  CHECK_ERROR( hipStreamDestroy(stPMLm) );
  CHECK_ERROR( hipStreamDestroy(stPMLp) );
  CHECK_ERROR( hipStreamDestroy(stX   ) ); 
  CHECK_ERROR( hipStreamDestroy(stP   ) ); 
  for(int i=0;i<NDev;i++) CHECK_ERROR( hipStreamDestroy(stDo[i]) );
  for(int i=0;i<NDev;i++) CHECK_ERROR( hipStreamDestroy(stDm[i]) );

  #ifdef MPI_ON
  if(even==0) for(int ixrag=ix; ixrag<ix+Nt-t0; ixrag++) DiamondRag::SendMPIp(subnode, ixrag);
  if(even==1) for(int ixrag=ix; ixrag<ix+Nt-t0; ixrag++) DiamondRag::SendMPIm(subnode, ixrag);
  #endif
}
inline void Window::Dtorres(int ix, int Nt, int t0, double disbal[NDev], bool isPMLs, bool isTFSF) {
  Dtorre<0>(ix,Nt,t0,disbal,isPMLs,isTFSF); //hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
  Dtorre<1>(ix,Nt,t0,disbal,isPMLs,isTFSF); //hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
}

#ifdef MPI_ON
MPI_Request reqSp, reqSm, reqRp, reqRm, reqSp_pml, reqSm_pml, reqRp_pml, reqRm_pml;
MPI_Status status;
int flag;
mpi_message Window::mes[8];
//#define BLOCK_SEND
#define MPI_TEST
//#define MPI_NUDGE
//#define USE_MPI_THREADING

#ifdef BLOCK_SEND
#define SendMPI(p,sz,tp,rnk,tag,world,req) MPI_Send(p,sz,tp,rnk,tag,world);
#define RecvMPI(p,sz,tp,rnk,tag,world,req) MPI_Recv(p,sz,tp,rnk,tag,world,&status);
#define doWait 0
#else
#ifndef USE_MPI_THREADING
#define WaitMPI(nreq,req,st) MPI_Wait(req,st)
#define SendMPI(p,sz,tp,rnk,tag,world,req,nreq) MPI_Isend(p,sz,tp,rnk,tag,world,req);
#define RecvMPI(p,sz,tp,rnk,tag,world,req,nreq) MPI_Irecv(p,sz,tp,rnk,tag,world,req);
#else
#define WaitMPI(nreq,req,st) { mpi_message* mes = &window.mes[nreq]; \
       int s=pthread_join(mes->mpith,0); if(s!=0) printf("node %d: Error joining thread %ld retcode=%d\n",window.node,mes->mpith,s); }
static void* send_func(void* args){
  mpi_message *mes = (mpi_message*)args;
  MPI_Send(mes->buf,mes->count,mes->datatype,mes->dest,mes->tag,mes->comm);
  return 0;
}
static void* recv_func(void* args){
  mpi_message *mes = (mpi_message*)args;
  MPI_Status stat;
  MPI_Recv(mes->buf,mes->count,mes->datatype,mes->dest,mes->tag,mes->comm,&stat);
  return 0;
}
#define SendMPI(p,sz,tp,rnk,tag,world,req,nreq) {mpi_message* mes = &window.mes[nreq]; mes->set(p,sz,tp,rnk,tag,world); \
      if(pthread_create(&mes->mpith,0,send_func,(void*)mes)!=0) {printf("Error: cannot create thread for MPI_send %d node=%d\n",nreq,window.node); MPI_Abort(MPI_COMM_WORLD, 1);};}
#define RecvMPI(p,sz,tp,rnk,tag,world,req,nreq) {mpi_message* mes = &window.mes[nreq]; mes->set(p,sz,tp,rnk,tag,world); \
      if(pthread_create(&mes->mpith,0,recv_func,(void*)mes)!=0) {printf("Error: cannot create thread for MPI_recv %d node=%d\n",nreq,window.node); MPI_Abort(MPI_COMM_WORLD, 1);};}
#endif//USE_MPI_THREADING
#define doWait 1
#endif
#endif// MPI_ON
int calcStep(){
//  CHECK_ERROR( hipDeviceSetSharedMemConfig ( hipSharedMemBankSizeEightByte ) );
  if(parsHost.iStep==0) printf("Starting...\n");
  cuTimer t0;
  int torreNum=0;
  CHECK_ERROR(hipDeviceSynchronize());
  #ifdef TEST_RATE
  for(int ix=Ns-Ntime; ix>0; ix--) {
//    printf("ix=%d\n",ix);
    torreD0<<<Na-2,Nv>>>(ix, 1, Ntime, 0); hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
    torreD1<<<Na-2,Nv>>>(ix, 1, Ntime, 0); hipDeviceSynchronize(); CHECK_ERROR( hipGetLastError() );
    torreNum++;
  }
  #else
  Window window; window.prepare();
  int node_shift=0; for(int inode=0; inode<window.node; inode++) node_shift+= mapNodeSize[inode]; node_shift-= Ns*window.node;
  int nsize=mapNodeSize[window.node]; int nL=node_shift; int nR=nL+nsize;
  #ifdef MPI_ON
  if(parsHost.iStep==0) {
    int wleftP=nR-Ns;
    int wleftM=nL;
    if(window.node!=window.Nprocs-1) {
      DEBUG_MPI(("Recv P (node %d) wleft=%d\n", window.node, wleftP));
      //MPI_Isend(&window.data    [wleftP*Na   ], Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node+1, 2+0, MPI_COMM_WORLD, &reqSp);
      //MPI_Isend(&window.dataPMLa[wleftP*Npmly], Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node+1, 2+1, MPI_COMM_WORLD, &reqSp_pml);
      #ifndef BLOCK_SEND
      int doSR=1;
      #ifdef MPI_TEST
      doSR=0;
      #endif
      RecvMPI(&window.data    [wleftP*Na   ], doSR*Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node+1, 2+0, MPI_COMM_WORLD, &reqRp    , 2);
      RecvMPI(&window.dataPMLa[wleftP*Npmly], doSR*Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node+1, 2+1, MPI_COMM_WORLD, &reqRp_pml, 6);
      #endif
    }
    if(window.node!=0              ) {
      //DEBUG_MPI(("Send&Recv M (node %d) wleft=%d\n", window.node, wleftM));
      //MPI_Isend(&window.data    [wleftM*Na   ], Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node-1, 2-2, MPI_COMM_WORLD, &reqSm);
      //MPI_Isend(&window.dataPMLa[wleftM*Npmly], Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node-1, 2-1, MPI_COMM_WORLD, &reqSm_pml);
      //MPI_Irecv(&window.data    [wleftM*Na   ], Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node-1, 2+0, MPI_COMM_WORLD, &reqRm);
      //MPI_Irecv(&window.dataPMLa[wleftM*Npmly], Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node-1, 2+1, MPI_COMM_WORLD, &reqRm_pml);
    }
  }
  #endif
  while(window.w0+Ns>=0) {
//    window.Memcopy();
    #ifdef MPI_ON
    #ifdef BLOCK_SEND
    if( !(parsHost.wleft>=nR && window.node!=window.Nprocs-1 || parsHost.wleft<nL-Ns && window.node!=0) ) {
      if(parsHost.wleft==nR-1     && window.node!=window.Nprocs-1) { 
        DEBUG_MPI(("bl Recv P(%d) (node %d) wleft=%d tag=%d\n", nR-Ns, window.node, parsHost.wleft, parsHost.iStep+0));
        RecvMPI(&window.data    [(nR-Ns)*Na   ], Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node+1, 2+(parsHost.iStep+0)*2+0, MPI_COMM_WORLD, &reqRp    , 2);
        RecvMPI(&window.dataPMLa[(nR-Ns)*Npmly], Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node+1, 2+(parsHost.iStep+0)*2+1, MPI_COMM_WORLD, &reqRp_pml, 6);
        DEBUG_MPI(("Ok Recv P(%d) (node %d) wleft=%d tag=%d\n", nR-Ns, window.node, parsHost.wleft, parsHost.iStep+0));
      }
      //if(parsHost.wleft==nR-Ns-Ns && window.node!=window.Nprocs-1) {
      if(parsHost.wleft==nL+Ns && window.node!=window.Nprocs-1) {
        DEBUG_MPI(("bl Send P(%d) (node %d) wleft=%d tag=%d\n", nR-Ns, window.node, parsHost.wleft, parsHost.iStep+1));
        SendMPI(&window.data    [(nR-Ns)*Na   ], Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node+1, 2+(parsHost.iStep+1)*2+0, MPI_COMM_WORLD, &reqSp     ,0);
        SendMPI(&window.dataPMLa[(nR-Ns)*Npmly], Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node+1, 2+(parsHost.iStep+1)*2+1, MPI_COMM_WORLD, &reqSp_pml, 4);
        DEBUG_MPI(("Ok Send P(%d) (node %d) wleft=%d tag=%d\n", nR-Ns, window.node, parsHost.wleft, parsHost.iStep+1));
      }
      if(parsHost.wleft==nL+Ns  && window.node!=0               && parsHost.iStep!=0) { 
        DEBUG_MPI(("bl Recv M(%d) (node %d) wleft=%d tag=%d\n", nL, window.node, parsHost.wleft, parsHost.iStep+0));
        RecvMPI(&window.data    [ nL    *Na   ], Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node-1, 2+(parsHost.iStep+0)*2+0, MPI_COMM_WORLD, &reqRm    , 3);
        RecvMPI(&window.dataPMLa[ nL    *Npmly], Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node-1, 2+(parsHost.iStep+0)*2+1, MPI_COMM_WORLD, &reqRm_pml, 7);
        DEBUG_MPI(("Ok Recv M(%d) (node %d) wleft=%d tag=%d\n", nL, window.node, parsHost.wleft, parsHost.iStep-1));
      }
      window.calcDtorres(nL,nR, parsHost.wleft<nL && window.node!=0, parsHost.wleft>=nR-Ns && window.node!=window.Nprocs-1);
      if(parsHost.wleft==nL-Ns  && window.node!=0              ) {
        DEBUG_MPI(("bl Send M(%d) (node %d) wleft=%d tag=%d\n", nL, window.node, parsHost.wleft, parsHost.iStep+0));
        SendMPI(&window.data    [ nL    *Na   ], Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node-1, 2+(parsHost.iStep+0)*2+0, MPI_COMM_WORLD, &reqSm    , 1);
        SendMPI(&window.dataPMLa[ nL    *Npmly], Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node-1, 2+(parsHost.iStep+0)*2+1, MPI_COMM_WORLD, &reqSm_pml, 5);
        DEBUG_MPI(("Ok Send M(%d) (node %d) wleft=%d tag=%d\n", nL, window.node, parsHost.wleft, parsHost.iStep+0));
      }
    }
    #else//BLOCK_SEND not def
    if( true /*!(parsHost.wleft>=nR && window.node!=window.Nprocs-1 || parsHost.wleft<nL-Ns && window.node!=0)*/ ) {
      bool doSend[2] = {1,1}; bool doRecv[2] = {1,1};
      #ifdef MPI_TEST
      if(parsHost.iStep  -window.node<0) { doSend[0]=0; doSend[1]=0; }
      if(parsHost.iStep+1-window.node<0) { doRecv[0]=0; doRecv[1]=0; }
      #endif
      if(doWait && parsHost.wleft==nR+(Ns-Ntime-1)   ) {
        DEBUG_MPI(("waiting P (node %d) wleft=%d\n", window.node, parsHost.wleft)); 
        if(window.node!=window.Nprocs-1                     ) { WaitMPI(2,&reqRp, &status);WaitMPI(6,&reqRp_pml, &status); }
        if(window.node!=0               && parsHost.iStep!=0) { WaitMPI(1,&reqSm, &status);WaitMPI(5,&reqSm_pml, &status); }
      }
      if(parsHost.wleft==nR-Ns-Ns-1 && window.node!=window.Nprocs-1) {
        DEBUG_MPI(("Send&Recv P(%d) (node %d) wleft=%d\n", parsHost.wleft+Ns, window.node, parsHost.wleft));
        SendMPI(&window.data    [(nR-Ns)*Na   ], doSend[1]*Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node+1, 2+(parsHost.iStep+1)*2+0, MPI_COMM_WORLD, &reqSp    ,0);
        SendMPI(&window.dataPMLa[(nR-Ns)*Npmly], doSend[1]*Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node+1, 2+(parsHost.iStep+1)*2+1, MPI_COMM_WORLD, &reqSp_pml,4);
        RecvMPI(&window.data    [(nR-Ns)*Na   ], doRecv[1]*Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node+1, 2+(parsHost.iStep+1)*2+0, MPI_COMM_WORLD, &reqRp    ,2);
        RecvMPI(&window.dataPMLa[(nR-Ns)*Npmly], doRecv[1]*Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node+1, 2+(parsHost.iStep+1)*2+1, MPI_COMM_WORLD, &reqRp_pml,6);
      }
      if(doWait && parsHost.wleft==nL+Ns+(Ns-Ntime-1)   && parsHost.iStep!=0) { 
        DEBUG_MPI(("waiting M (node %d) wleft=%d\n", window.node, parsHost.wleft)); 
        if(window.node!=0              ) { WaitMPI(3,&reqRm, &status);WaitMPI(7,&reqRm_pml, &status); }
        if(window.node!=window.Nprocs-1) { WaitMPI(0,&reqSp, &status);WaitMPI(4,&reqSp_pml, &status); }
      }
      #ifdef MPI_NUDGE
      if(doWait && (parsHost.wleft+Ns)%1==0) {
        if(parsHost.iStep!=0 && window.node!=window.Nprocs-1) { DEBUG_MPI(("testing sendP (node %d) wleft=%d\n", window.node, parsHost.wleft)); MPI_Test(&reqSp, &flag, &status);MPI_Test(&reqSp_pml, &flag, &status); }
        if(                     window.node!=window.Nprocs-1) { DEBUG_MPI(("testing recvP (node %d) wleft=%d\n", window.node, parsHost.wleft)); MPI_Test(&reqRp, &flag, &status);MPI_Test(&reqRp_pml, &flag, &status); }
        if(parsHost.iStep!=0 && window.node!=0              ) { DEBUG_MPI(("testing sendM (node %d) wleft=%d\n", window.node, parsHost.wleft)); MPI_Test(&reqSm, &flag, &status);MPI_Test(&reqSm_pml, &flag, &status); }
        if(parsHost.iStep!=0 && window.node!=0              ) { DEBUG_MPI(("testing recvM (node %d) wleft=%d\n", window.node, parsHost.wleft)); MPI_Test(&reqRm, &flag, &status);MPI_Test(&reqRm_pml, &flag, &status); }
      }
      #endif
      #ifdef MPI_TEST
      if(parsHost.iStep-window.node>0)
      #endif
             #pragma omp task
      window.calcDtorres(nL,nR, parsHost.wleft<nL && window.node!=0, parsHost.wleft>=nR-Ns && window.node!=window.Nprocs-1);
             #pragma omp taskwait

      if(parsHost.wleft==nL-Ns-1  && window.node!=0              ) {
        DEBUG_MPI(("Send&Recv M(%d) (node %d) wleft=%d\n", parsHost.wleft+Ns+1, window.node, parsHost.wleft));
        SendMPI(&window.data    [ nL    *Na   ], doSend[0]*Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node-1, 2+(parsHost.iStep  )*2+0, MPI_COMM_WORLD, &reqSm    ,1);
        SendMPI(&window.dataPMLa[ nL    *Npmly], doSend[0]*Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node-1, 2+(parsHost.iStep  )*2+1, MPI_COMM_WORLD, &reqSm_pml,5);
        RecvMPI(&window.data    [ nL    *Na   ], doRecv[0]*Ns*Na   *sizeof(DiamondRag   )/sizeof(ftype), MPI_FTYPE, window.node-1, 2+(parsHost.iStep+1)*2+0, MPI_COMM_WORLD, &reqRm,    3);
        RecvMPI(&window.dataPMLa[ nL    *Npmly], doRecv[0]*Ns*Npmly*sizeof(DiamondRagPML)/sizeof(ftype), MPI_FTYPE, window.node-1, 2+(parsHost.iStep+1)*2+1, MPI_COMM_WORLD, &reqRm_pml,7);
      }
    }
    #endif//BLOCK_SEND
    #else//MPI_ON not def
    window.calcDtorres();
    #endif//MPI_ON
    window.synchronize();
  }
  parsHost.drop.dump();
  #ifndef MPI_TEST
  if(0 && parsHost.iStep%(10*window.Nprocs)==0) parsHost.drop.sync();
  #endif
//    printf("ix=%d\n",ix);
/*    int zones[] = {0, Npmlx/2, tfsfSm/dx/NDT-2, tfsfSp/dx/NDT+2, Ns-Npmlx/2, Ns}; int izon=0;
    Dtorres(max(ix,zones[izon]), min(Ntime,zones[izon+1]-ix), max(zones[izon]-ix,0), true );
    
    izon++;
    Dtorres(max(ix,zones[izon]), min(Ntime,zones[izon+1]-ix), max(zones[izon]-ix,0), false);
    izon++;
    Dtorres(max(ix,zones[izon]), min(Ntime,zones[izon+1]-ix), max(zones[izon]-ix,0), false, ((parsHost.iStep+1)*Ntime*dt<shotpoint.tStop)?true:false);
    izon++;
    Dtorres(max(ix,zones[izon]), min(Ntime,zones[izon+1]-ix), max(zones[izon]-ix,0), false);

    izon++;
    Dtorres(max(ix,zones[izon]), min(Ntime,zones[izon+1]-ix), max(zones[izon]-ix,0), true );*/
  #endif

  double calcTime=t0.gettime();
  double yee_cells = 0;
  double overhead=0;
  #ifndef TEST_RATE
  yee_cells = NDT*NDT*Ntime*(unsigned long long)(Nv*Na)*Np;
  overhead = window.RAMcopytime/window.GPUcalctime;
  printf("Step %d /node %d/: Time %9.09f ms |overhead %3.03f%% |disbalance ", parsHost.iStep, window.node, calcTime, 100*overhead);
  for(int idev=0;idev<NDev;idev++) printf("%3.03f%% ", 100*window.disbal[idev]/window.GPUcalctime);
  printf("|rate %9.09f GYee_cells/sec |isTFSF=%d \n", 1.e-9*yee_cells/(calcTime*1.e-3), (parsHost.iStep+1)*Ntime*dt<shotpoint.tStop );
  #else
  yee_cells = NDT*NDT*Ntime*(unsigned long long)(Nv*(Na-2))*torreNum;
  printf("Step %d: Time %9.09f ms |overhead %3.03f%% |rate %9.09f %d %d %d %d (GYee cells/sec,Nx,Ny,Nz,Ntime) |isTFSF=%d \n", parsHost.iStep, calcTime, 100*overhead, 1.e-9*yee_cells/(calcTime*1.e-3), Nx,Ny,Nz,Ntime, (parsHost.iStep+1)*Ntime*dt<shotpoint.tStop );
  #endif
  #ifdef MPI_ON
  double AllCalcTime;
  MPI_Reduce(&calcTime, &AllCalcTime, 1, MPI_DOUBLE, MPI_MAX, 0, MPI_COMM_WORLD);
  if(window.node==0 && 0) printf("===(%3d)===AllCalcTime %9.09f sec |rate %9.09f GYee_cells/sec\n", parsHost.iStep, AllCalcTime*1e-3, 1.e-9*yee_cells/(AllCalcTime*1.e-3) );
  #endif
  parsHost.iStep++;
  copy2dev(parsHost, pars);
  return 0; 
}
