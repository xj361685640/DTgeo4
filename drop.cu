#define WSIZE 32
const int Nwarps=Nz/WSIZE;
extern __constant__ uint32_t drop_cells[Ns*NDT*Nwarps];
struct SeismoDrops {
  ftype* channel[9]; // SxyzTxyzVxyz
  ftype* channelHost[9]; // SxyzTxyzVxyz
  ftype** channelAddr;
  ftype* channelAddrHost[9];
  ptrdiff_t offset[9];
  uint32_t *drop_cellsHost;
  #ifdef MPI_ON
  MPI_File file[9]; MPI_Status status; MPI_Info info;
  #else
  FILE* file[9];
  #endif
  static const int channelHostLength=Ntime*Np*NDT*100; // future think about size
  static const int channelDevLength=Ntime*Ns*NDT*100; // future think about size
  int node,Nprocs;
  std::string* dir;
  void init() {
    node=0; Nprocs=1;
    #ifdef MPI_ON
    MPI_Comm_rank (MPI_COMM_WORLD, &node);
    MPI_Comm_size (MPI_COMM_WORLD, &Nprocs);
    #endif
    printf("Size of drop_cells =%.2fKB\n", sizeof(uint32_t)*Ns*NDT*Nwarps/1024.);
    
    //get drop_cells from aivView, streamSys
    drop_cellsHost = new uint32_t[Np*NDT*Nwarps]; memset(drop_cellsHost, 0, sizeof(uint32_t)*Np*NDT*Nwarps);
    //     ____!!!___first warp is for PML___!!!___   NOPE, IT IS WRONG?? CHECK LATER
    //for(int ix=0; ix<NDT*Np; ix++) {drop_cellsHost[ix*Nwarps+Nz/32/2]=1/*65*/; drop_cellsHost[ix*Nwarps+Nz/32/2-1]=1; drop_cellsHost[ix*Nwarps+Nz/32/2+1]=1;}
    #if 1 
    for(int ix=0; ix<NDT*Np; ix+=3) for(int sline=0; sline<Nz/2-Npmlz/2; sline+=6) {
      int line=Nz/2+sline; int th_id=line; drop_cellsHost[ix*Nwarps+(th_id)/32]|=((uint64_t)1)<<(th_id%32); 
          line=Nz/2-sline;     th_id=line; drop_cellsHost[ix*Nwarps+(th_id)/32]|=((uint64_t)1)<<(th_id%32);
    } 
    for(int iw=0; iw<Nwarps; iw++) drop_cellsHost[Np*NDT/2*Nwarps+iw]=(((uint64_t)1)<<32)-1;
    #endif
    
    //CHECK_ERROR( hipMalloc((void**)&drop_cells, sizeof(uint32_t)*Np*Nwarps) );
    //CHECK_ERROR( hipMemcpy(drop_cells, drop_cellsHost, sizeof(uint32_t)*Np*Nwarps, hipMemcpyHostToDevice) );
    
    CHECK_ERROR( hipMalloc((void**)&channelAddr, sizeof(ftype*)*9) );
    for(int fld=0; fld<9; fld++){
      channelHost[fld] = new ftype[channelHostLength];
      offset[fld]=0;
      CHECK_ERROR( hipMalloc((void**)&channel[fld], sizeof(ftype)*channelDevLength) ); // future think about size
      channelAddrHost[fld] = channel[fld];
    }
    CHECK_ERROR( hipMemcpy(channelAddr, channelAddrHost, sizeof(ftype*)*9, hipMemcpyHostToDevice) ); 
    char sf[256];
    
    #ifdef MPI_ON
    MPI_Info_create( &info );
    //MPI_Info_set(info, "romio_ds_write", "disable");
    //MPI_Info_set(info, "romio_ds_read",  "disable");
    for(int fn=0; fn<9; fn++) {
      switch(fn) {
        case 0: sprintf(sf,"%s/Sx.arr",dir->c_str()); break;
        case 1: sprintf(sf,"%s/Sz.arr",dir->c_str()); break;
        case 2: sprintf(sf,"%s/Sy.arr",dir->c_str()); break;
        case 3: sprintf(sf,"%s/Tx.arr",dir->c_str()); break;
        case 4: sprintf(sf,"%s/Tz.arr",dir->c_str()); break;
        case 5: sprintf(sf,"%s/Ty.arr",dir->c_str()); break;
        case 6: sprintf(sf,"%s/Vx.arr",dir->c_str()); break;
        case 7: sprintf(sf,"%s/Vz.arr",dir->c_str()); break;
        case 8: sprintf(sf,"%s/Vy.arr",dir->c_str()); break;
        default: printf("Unknown file number\n"); exit(-1);
      }
      //MPI_File_open( MPI_COMM_WORLD, sf, MPI_MODE_CREATE | MPI_MODE_DELETE_ON_CLOSE | MPI_MODE_WRONLY, MPI_INFO_NULL, &file[fn] );
      //MPI_File_close( &file[fn] );
      if(node==0) remove(sf);
      MPI_Barrier(MPI_COMM_WORLD);
      MPI_File_open( MPI_COMM_WORLD, sf, MPI_MODE_CREATE | MPI_MODE_EXCL            | MPI_MODE_WRONLY, info         , &file[fn] );
      MPI_File_set_atomicity(file[fn], true);
    }
    if(node==0) for(int i=0;i<9;i++) {
      uint32_t val = Np*NDT; MPI_File_write_shared(file[i], &val, 1, MPI_UNSIGNED, &status);
               val = Nwarps; MPI_File_write_shared(file[i], &val, 1, MPI_UNSIGNED, &status);
      MPI_File_write_shared(file[i], drop_cellsHost, Np*NDT*Nwarps, MPI_UNSIGNED, &status);
    }
    #else
    sprintf(sf,"%s/Sx.arr",dir->c_str()); file[0] = fopen(sf, "w"); if(file[0]==NULL) perror("Cannot open file for dump\n");
    sprintf(sf,"%s/Sz.arr",dir->c_str()); file[1] = fopen(sf, "w"); if(file[1]==NULL) perror("Cannot open file for dump\n");
    sprintf(sf,"%s/Sy.arr",dir->c_str()); file[2] = fopen(sf, "w"); if(file[2]==NULL) perror("Cannot open file for dump\n");
    sprintf(sf,"%s/Tx.arr",dir->c_str()); file[3] = fopen(sf, "w"); if(file[3]==NULL) perror("Cannot open file for dump\n");
    sprintf(sf,"%s/Tz.arr",dir->c_str()); file[4] = fopen(sf, "w"); if(file[4]==NULL) perror("Cannot open file for dump\n");
    sprintf(sf,"%s/Ty.arr",dir->c_str()); file[5] = fopen(sf, "w"); if(file[5]==NULL) perror("Cannot open file for dump\n");
    sprintf(sf,"%s/Vx.arr",dir->c_str()); file[6] = fopen(sf, "w"); if(file[6]==NULL) perror("Cannot open file for dump\n");
    sprintf(sf,"%s/Vz.arr",dir->c_str()); file[7] = fopen(sf, "w"); if(file[7]==NULL) perror("Cannot open file for dump\n");
    sprintf(sf,"%s/Vy.arr",dir->c_str()); file[8] = fopen(sf, "w"); if(file[8]==NULL) perror("Cannot open file for dump\n");
    for(int i=0;i<9;i++) {
      uint32_t val = Np*NDT; fwrite(&val, sizeof(uint32_t), 1, file[i]);
               val = Nwarps; fwrite(&val, sizeof(uint32_t), 1, file[i]);
      fwrite(drop_cellsHost, sizeof(uint32_t), Np*NDT*Nwarps, file[i]);
    }
    #endif
  }
  void copy_drop_cells(const int ixdev, const int ixhost, hipStream_t& stream){
    DEBUG_PRINT(("copy drop cells HtoD ixdev=%d ixhost=%d \\ yes %d\n", ixdev, ixhost, ixhost<Np && ixhost>=0));
    if(ixhost<Np && ixhost>=0) CHECK_ERROR( hipMemcpyToSymbolAsync(HIP_SYMBOL(drop_cells), &drop_cellsHost[ixhost*NDT*Nwarps], sizeof(uint32_t)*NDT*Nwarps, sizeof(uint32_t)*ixdev*NDT*Nwarps, hipMemcpyHostToDevice, stream) );
  }
  void save(hipStream_t& stream){
    DEBUG_PRINT(("save channel from device to host\n"));
    CHECK_ERROR( hipMemcpyAsync(channelAddrHost, channelAddr, sizeof(ftype*)*9, hipMemcpyDeviceToHost, stream) );
    CHECK_ERROR( hipStreamSynchronize(stream) );
    for(int i=0;i<9;i++) if(offset[i]+channelAddrHost[i]-channel[i] >= channelHostLength) { printf("Length of Host buffer channel %d is exceeded (to fix increase channelHostLength value)\n", i); exit(-1); }
    for(int i=0;i<9;i++) CHECK_ERROR( hipMemcpyAsync((channelHost[i]+offset[i]), channel[i], (channelAddrHost[i]-channel[i])*sizeof(ftype), hipMemcpyDeviceToHost, stream) );
    CHECK_ERROR( hipStreamSynchronize(stream) );
    for(int i=0;i<9;i++) DEBUG_PRINT(("channel%d = %p, channelHost%d = %p, channelAddrHost%d = %p\n", i,channel[i], i,channelHost[i], i,channelAddrHost[i]));
    for(int i=0;i<9;i++) offset[i]+= (channelAddrHost[i]-channel[i]);
    for(int i=0;i<9;i++) DEBUG_PRINT(("new offset%d = %d\n", i,offset[i]));
    //---reset---
    for(int fld=0; fld<9; fld++) channelAddrHost[fld] = channel[fld];
    CHECK_ERROR( hipMemcpyAsync(channelAddr, channelAddrHost, sizeof(ftype*)*9, hipMemcpyHostToDevice, stream) );
    //int i=0; for(ftype* p=channelHost[i]; p!= channelHost[i]+offset[i]; p++) printf("buffer data p=%p val=%g\n", p, *p);
  }
  void dump(){
    DEBUG_PRINT(("dump channel data to file node=%d\n",node));
    #ifdef MPI_ON
    for(int i=0;i<9;i++) MPI_File_write_shared(file[i], channelHost[i], offset[i], MPI_FLOAT, &status);
    #else
    for(int i=0;i<9;i++) fwrite(channelHost[i], sizeof(ftype), offset[i], file[i]);
    #endif
    for(int i=0;i<9;i++) offset[i] = 0;
    DEBUG_PRINT(("ok dump channel data to file node=%d\n",node));
  }
  void sync(){
    #ifdef MPI_ON
    MPI_Barrier(MPI_COMM_WORLD);
    for(int i=0;i<9;i++) MPI_File_sync(file[i]);
    #else
//    for(int i=0;i<9;i++) fflush(file[i]);
    #endif
  }
};
